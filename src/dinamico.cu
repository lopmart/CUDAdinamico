#include "hip/hip_runtime.h"
%%writefile dinamico.cu
#include <iostream>
#include <hip/hip_runtime.h>

// Kernel secundario que se lanzará de manera dinámica
__global__ void kernelSecundario() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hilo %d ejecutando kernel secundario\n", idx);
}

// Kernel principal que lanza el kernel secundario
__global__ void kernelPrincipal() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hilo %d ejecutando kernel principal\n", idx);

    // Lanzamiento dinámico de un 
    //nuevo kernel desde dentro del kernel principal
    if (idx == 0) {
        kernelSecundario<<<1, 10>>>();
    }
}

int main() {
    // Lanzamiento del kernel principal
    kernelPrincipal<<<1, 10>>>( );
    
    // Sincronización
    hipDeviceSynchronize();
    
    return 0;
}
